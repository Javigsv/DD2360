#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#define NUM_ITERATIONS 100
#define SEED 787
#define MARGIN 1e-6

int NUM_PARTICLES;
int BLOCK_SIZE;

typedef struct Particle {
    float3 position;
    float3 velocity;
} Particle;

float floatRand(float max) {
    return ((float)rand()/RAND_MAX)*max;
}

void initParticle(Particle* p) {
    float x = floatRand(5.0);
    float y = floatRand(5.0);
    float z = floatRand(5.0);
    float vx = floatRand(5.0);
    float vy = floatRand(5.0);
    float vz = floatRand(5.0);
    p->position = make_float3(x, y, z);
    p->velocity = make_float3(vx, vy, vz);
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}   

__host__ __device__ float gen_random(int seed, int p, int i, int NUM_PARTICLES) {
    return (seed*p+i) % NUM_PARTICLES;
}

void timeStepCPU(Particle* p, float dt, int iter) {
    float x;
    float y;
    float z;
    float3 pos;
    float3 vel;
    float randomV;
    for (int i = 0; i < NUM_PARTICLES; i++) {
        pos = p[i].position;
        x = pos.x;
        y = pos.y;
        z = pos.z;
        randomV = gen_random(SEED, i, iter, NUM_PARTICLES);
        p[i].velocity.x = randomV*0.2;
        p[i].velocity.y = randomV*0.5;
        p[i].velocity.z = randomV*0.3;
        vel = p[i].velocity;
        p[i].position = make_float3(x+vel.x*dt, y+vel.y*dt, z+vel.z*dt);
    }
}

__device__ void timeStepGPU(Particle* p, float dt, int iter, int NUM_PARTICLES) {
    float x;
    float y;
    float z;
    float3 pos;
    float3 vel;
    float randomV;
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < NUM_PARTICLES) 
    {
        pos = p[id].position;
        x = pos.x;
        y = pos.y;
        z = pos.z;
        randomV = gen_random(SEED, id, iter, NUM_PARTICLES);
        p[id].velocity.x = randomV*0.2;
        p[id].velocity.y = randomV*0.5;
        p[id].velocity.z = randomV*0.3;
        vel = p[id].velocity;
        p[id].position = make_float3(x+vel.x*dt, y+vel.y*dt, z+vel.z*dt);
    }
}

__global__ void executeGPU(Particle* p, float dt, int NUM_PARTICLES) {
    for(int i = 0; i < NUM_ITERATIONS; i++) {
        timeStepGPU(p, dt, i, NUM_PARTICLES);
    }
}

int main(int argc, char* argv[]){
    NUM_PARTICLES = atoi(argv[1]);
    BLOCK_SIZE = atoi(argv[2]);
    printf("%d particles\nBlock Size = %d\n", NUM_PARTICLES, BLOCK_SIZE);
    Particle* particles = (Particle*)malloc(NUM_PARTICLES*sizeof(Particle));
    int i = 0;
    Particle* particlesGPU;
    Particle* solutionGPU = (Particle*)malloc(NUM_PARTICLES*sizeof(Particle));

    double t1;
    double t2;
    double timeCPU;
    double timeGPU;

    srand((unsigned) time(NULL)); 

    t1 = cpuSecond();
    hipMalloc(&particlesGPU, NUM_PARTICLES*sizeof(Particle));
    hipMemcpy(particlesGPU, particles, NUM_PARTICLES*sizeof(Particle), hipMemcpyHostToDevice);
    t2 = cpuSecond();
    timeGPU = t2- t1;

    //printf("Init: (%f, %f, %f)\n", particles[0].position.x, particles[0].position.y, particles[0].position.z);
    //CPU
    printf("Computing in the CPU...\n");
    t1 = cpuSecond();
    for(i = 0; i < NUM_ITERATIONS; i++) {
        timeStepCPU(particles, 1, i);
    }
    t2 = cpuSecond();
    timeCPU = t2-t1;
    //printf("%d: (%f, %f, %f)\n", i, particles[0].position.x, particles[0].position.y, particles[0].position.z);
    printf("Done! %f\n", timeCPU);

    //GPU
    printf("Computing in the GPU...\n");
    t1 = cpuSecond();
    executeGPU<<<(NUM_PARTICLES+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(particlesGPU, 1, NUM_PARTICLES);
    hipDeviceSynchronize();
    hipMemcpy(solutionGPU, particlesGPU, NUM_PARTICLES*sizeof(Particle), hipMemcpyDeviceToHost);
    t2 = cpuSecond();
    timeGPU += t2-t1;
    //printf("%d: (%f, %f, %f)\n", i, solutionGPU[0].position.x, solutionGPU[0].position.y, solutionGPU[0].position.z);
    printf("Done! %f\n", timeGPU);

    printf("Comparing results...\n");
    for(i = 0; i < NUM_PARTICLES; i++) {
        float xCPU = particles[i].position.x;
        float yCPU = particles[i].position.y;
        float zCPU = particles[i].position.z;
        float xGPU = solutionGPU[i].position.x;
        float yGPU = solutionGPU[i].position.y;
        float zGPU = solutionGPU[i].position.z;
        if(fabs(xCPU - xGPU) > MARGIN | fabs(yCPU - yGPU) > MARGIN | fabs(zCPU - zGPU) > MARGIN) {
            printf("CPU: (%f, %f, %f)\n", xCPU, yCPU, zCPU);
            printf("GPU: (%f, %f, %f)\n", xGPU, yGPU, zGPU);
            printf("Something is bad %d\n", i);
            exit(0);
        }
    }
    printf("Everything seems fine\n");
}
